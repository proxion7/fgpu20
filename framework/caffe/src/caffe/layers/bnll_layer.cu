#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bnll_layer.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

const float kBNLL_THRESHOLD = 50.;

#ifndef USE_FGPU
template <typename Dtype>
__global__ void BNLLForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ?
        in[index] + log(1. + exp(-in[index])) :
        log(1. + exp(in[index]));
  }
}


template <typename Dtype>
__global__ void BNLLBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype expval = exp(min(in_data[index], Dtype(kBNLL_THRESHOLD)));
    out_diff[index] = in_diff[index] * expval / (expval + 1.);
  }
}

#else
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(BNLLForward, const int n, const Dtype* in, Dtype* out) {

  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, n, _blockIdx, _gridDim) {
      Dtype in_val = FGPU_COLOR_LOAD(ctx, &in[index]);

      FGPU_COLOR_STORE(ctx, &out[index], in_val > 0 ?
          in_val + log(1. + exp(-in_val)) :
          log(1. + exp(in_val)));
    }
    
  }
}


template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(BNLLBackward, const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {

  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, n, _blockIdx, _gridDim) {
      Dtype expval = exp(min(FGPU_COLOR_LOAD(ctx, &in_data[index]), Dtype(kBNLL_THRESHOLD)));
      FGPU_COLOR_STORE(ctx, &out_diff[index], 
              FGPU_COLOR_LOAD(ctx, &in_diff[index]) * expval / (expval + 1.));
    }
 
  }
}

#endif

template <typename Dtype>
void BNLLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
#ifndef USE_FGPU
  // NOLINT_NEXT_LINE(whitespace/operators)
  BNLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
#else
  FGPU_CHECK(FGPU_LAUNCH_KERNEL(BNLLForward<Dtype>,
      CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0,
      count, bottom_data, top_data));

#endif
}


template <typename Dtype>
void BNLLLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
#ifndef USE_FGPU
    // NOLINT_NEXT_LINE(whitespace/operators)
    BNLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(BNLLBackward<Dtype>,
        CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0,
        count, top_diff, bottom_data, bottom_diff));

#endif
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BNLLLayer);


}  // namespace caffe
