#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/concat_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

#ifndef USE_FGPU
template <typename Dtype>
__global__ void Concat(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_concat_size = concat_size * bottom_concat_axis;
    const int concat_num = index / total_concat_size;
    const int concat_index = index % total_concat_size;
    const int top_index = concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
    }
  }
}

#else
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(Concat, const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {

  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, nthreads, _blockIdx, _gridDim) {
      const int total_concat_size = concat_size * bottom_concat_axis;
      const int concat_num = index / total_concat_size;
      const int concat_index = index % total_concat_size;
      const int top_index = concat_index +
          (concat_num * top_concat_axis + offset_concat_axis) * concat_size;

      if (forward) {
        FGPU_COLOR_STORE(ctx, &out_data[top_index],
                FGPU_COLOR_LOAD(ctx, &in_data[index]));
      } else {
        FGPU_COLOR_STORE(ctx, &out_data[index],
                FGPU_COLOR_LOAD(ctx, &in_data[top_index]));
      }
    }

  } 
}

#endif

template <typename Dtype>
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (bottom.size() == 1) { return; }
  Dtype* top_data = top[0]->mutable_gpu_data();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int nthreads = bottom_concat_size * num_concats_;
#ifndef USE_FGPU
    Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
        top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(Concat<Dtype>,  // NOLINT_NEXT_LINE(whitespace/operators)
        CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0,
        nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
        top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data));

#endif
    offset_concat_axis += bottom_concat_axis;
  }
}

template <typename Dtype>
void ConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int i = 0; i < bottom.size(); ++i) {
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int nthreads = bottom_concat_size * num_concats_;
#ifndef USE_FGPU
      Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff);
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(Concat<Dtype>,  // NOLINT_NEXT_LINE(whitespace/operators)
          CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0,
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff));

#endif
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConcatLayer);

}  // namespace caffe
