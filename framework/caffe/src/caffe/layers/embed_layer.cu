#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/embed_layer.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/util/math_functions.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

#ifndef USE_FGPU

template <typename Dtype>
__global__ void EmbedForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int M, const int N, const int K,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    #ifdef DEBUG
        assert(index >= 0);
        assert(index < K);
        assert(static_cast<Dtype>(index) == bottom_data[n]);
    #endif
    const int weight_index = index * N + d;
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int M, const int N, const int K,
    Dtype* weight_diff);

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int M, const int N, const int K,
    Dtype* weight_diff) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = index * N + d;
    caffe_gpu_atomic_add(top_diff[top_index], weight_diff + weight_index);
  }
}

#else // USE_FGPU

template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(EmbedForward, const int nthreads, 
    const Dtype* bottom_data, const Dtype* weight, const int M, const int N, 
    const int K, Dtype* top_data) {
  
  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {
    CUDA_KERNEL_LOOP(top_index, nthreads, _blockIdx, _gridDim) {
      const int n = top_index / N;
      const int d = top_index % N;
      Dtype bdata = FGPU_COLOR_LOAD(ctx, &bottom_data[n]);
      const int index = static_cast<int>(bdata);
      #ifdef DEBUG
        assert(index >= 0);
        assert(index < K);
        assert(static_cast<Dtype>(index) == bdata);
      #endif
      const int weight_index = index * N + d;
      FGPU_COLOR_STORE(ctx, &top_data[top_index], 
        FGPU_COLOR_LOAD(ctx, &weight[weight_index]));
    }
  }
}

template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(EmbedBackward, const int nthreads,
    const Dtype* bottom_data, const Dtype* top_diff, const int M, 
    const int N, const int K, Dtype* weight_diff) {
    
  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {
    CUDA_KERNEL_LOOP(top_index, nthreads, _blockIdx, _gridDim) {
      const int n = top_index / N;
      const int d = top_index % N;
      Dtype bdata = FGPU_COLOR_LOAD(ctx, &bottom_data[n]);
      const int index = static_cast<int>(bdata);
      const int weight_index = index * N + d;
      caffe_gpu_atomic_add(FGPU_COLOR_LOAD(ctx, &top_diff[top_index]), 
        FGPU_COLOR_TRANSLATE_ADDR(ctx, &weight_diff[weight_index]));
    }
  }
}

#endif // USE_FGPU

template <typename Dtype>
void EmbedLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
#ifndef USE_FGPU
  EmbedForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight, M_, N_, K_, top_data);
#else
  FGPU_CHECK(FGPU_LAUNCH_KERNEL(EmbedForward<Dtype>, CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS, 0, count, bottom_data, weight, M_, N_, K_, top_data));
#endif
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, Dtype(1),
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), Dtype(1), top_data);
  }
}

template <typename Dtype>
void EmbedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  CHECK(!propagate_down[0]) << "Can't backpropagate to EmbedLayer input.";
  if (this->param_propagate_down_[0]) {
    const int top_count = top[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
#ifndef USE_FGPU
    EmbedBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS>>>(
        top_count, bottom_data, top_diff, M_, N_, K_, weight_diff);
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(EmbedBackward<Dtype>, CAFFE_GET_BLOCKS(top_count), 
        CAFFE_CUDA_NUM_THREADS, 0, top_count, bottom_data, top_diff, M_, N_, K_, 
        weight_diff));
#endif
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, Dtype(1), top_diff,
        bias_multiplier_.gpu_data(), Dtype(1), bias_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EmbedLayer);

}  // namespace caffe
