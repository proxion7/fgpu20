#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/prelu_layer.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

#ifndef USE_FGPU
// CUDA kernele for forward
template <typename Dtype>
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void PReLUBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * slope_data[c]);
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void PReLUParamBackward(const int n,
    const int rows, const int rowPitch, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= 0);
    for ( int k = 1; k < rows; k++ ) {
        out_diff[index] += in_diff[index + k*rowPitch]
           * in_data[index + k*rowPitch] * (in_data[index + k*rowPitch] <= 0);
    }
  }
}

#else // USE_FGPU
// CUDA kernele for forward
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(PReLUForward, const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data,
    const int div_factor) {
    
  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, n, _blockIdx, _gridDim) {
      int c = (index / dim) % channels / div_factor;
      Dtype in_val = FGPU_COLOR_LOAD(ctx, &in[index]);
      FGPU_COLOR_STORE(ctx, &out[index],
              in_val > 0 ? in_val : in_val * FGPU_COLOR_LOAD(ctx, &slope_data[c]));
    }

  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(PReLUBackward, const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const int div_factor) {
  
    
  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {
      
    CUDA_KERNEL_LOOP(index, n, _blockIdx, _gridDim) {
      int c = (index / dim) % channels / div_factor;
      Dtype in_data_val = FGPU_COLOR_LOAD(ctx, &in_data[index]);
      FGPU_COLOR_STORE(ctx, &out_diff[index],
              FGPU_COLOR_LOAD(ctx, &in_diff[index]) * ((in_data_val > 0)
          + (in_data_val <= 0) * FGPU_COLOR_LOAD(ctx, &slope_data[c])));
    }

  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(PReLUParamBackward, const int n,
    const int rows, const int rowPitch, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {

  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, n, _blockIdx, _gridDim) {
      Dtype in_data_val = FGPU_COLOR_LOAD(ctx, &in_data[index]);
      Dtype in_diff_val = FGPU_COLOR_LOAD(ctx, &in_diff[index]);
      Dtype *out_diff_addr = FGPU_COLOR_TRANSLATE_ADDR(ctx, &out_diff[index]);

      *out_diff_addr = in_diff_val * in_data_val * (in_data_val <= 0);
      for ( int k = 1; k < rows; k++ ) {
          in_data_val = FGPU_COLOR_LOAD(ctx, &in_data[index + k*rowPitch]);
          in_diff_val = FGPU_COLOR_LOAD(ctx, &in_diff[index + k*rowPitch]);

          *out_diff_addr += in_diff_val * in_data_val * (in_data_val <= 0);
      }
    }

  }
}

#endif

template <typename Dtype>
void PReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* slope_data = this->blobs_[0]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

#ifndef USE_FGPU
  // NOLINT_NEXT_LINE(whitespace/operators)
  PReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, slope_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
#else
  FGPU_CHECK(FGPU_LAUNCH_KERNEL(PReLUForward<Dtype>,
      CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0,
      count, channels, dim, bottom_data, top_data, slope_data, div_factor));
#endif
}

template <typename Dtype>
void PReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagate to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.
  if (this->param_propagate_down_[0]) {
    Dtype* slope_diff = this->blobs_[0]->mutable_gpu_diff();
    int cdim = channels * dim;

#ifndef USE_FGPU
    // compute element-wise diff
    // NOLINT_NEXT_LINE(whitespace/operators)
    PReLUParamBackward<Dtype><<<CAFFE_GET_BLOCKS(cdim),
      CAFFE_CUDA_NUM_THREADS>>>(
      cdim, bottom[0]->num(), top[0]->offset(1), top_diff ,
      bottom_data ,
      backward_buff_.mutable_gpu_diff());
    CUDA_POST_KERNEL_CHECK;
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(PReLUParamBackward<Dtype>,
      CAFFE_GET_BLOCKS(cdim), CAFFE_CUDA_NUM_THREADS, 0,
      cdim, bottom[0]->num(), top[0]->offset(1), top_diff ,
      bottom_data ,
      backward_buff_.mutable_gpu_diff()));
#endif
    if (channel_shared_) {
      Dtype dsum;
      caffe_gpu_dot<Dtype>(channels * dim, backward_buff_.gpu_diff(),
       multiplier_.gpu_data(), &dsum);
      caffe_gpu_add_scalar(this->blobs_[0]->count(), Dtype(dsum), slope_diff);
    } else {
      caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
        backward_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
        slope_diff);
    }
  }
  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = this->blobs_[0]->gpu_data();
    int div_factor = channel_shared_ ? channels : 1;

#ifndef USE_FGPU
    // NOLINT_NEXT_LINE(whitespace/operators)
    PReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data,
        div_factor);
    CUDA_POST_KERNEL_CHECK;
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(PReLUBackward<Dtype>,
        CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0,
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data,
        div_factor));
#endif
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(PReLULayer);


}  // namespace caffe
