#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/slice_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

#ifndef USE_FGPU
template <typename Dtype>
__global__ void Slice(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}
#else
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(Slice, const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
 
  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(index, nthreads, _blockIdx, _gridDim) {
      const int total_slice_size = slice_size * top_slice_axis;
      const int slice_num = index / total_slice_size;
      const int slice_index = index % total_slice_size;
      const int bottom_index = slice_index +
          (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
      if (forward) {
        FGPU_COLOR_STORE(ctx, &out_data[index], FGPU_COLOR_LOAD(ctx, &in_data[bottom_index]));
      } else {
        FGPU_COLOR_STORE(ctx, &out_data[bottom_index], FGPU_COLOR_LOAD(ctx, &in_data[index]));
      }
    }

  } 
}
#endif

template <typename Dtype>
void SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (top.size() == 1) { return; }
  int offset_slice_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = true;
  for (int i = 0; i < top.size(); ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
#ifndef USE_FGPU
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
#else 
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(Slice<Dtype>,  // NOLINT_NEXT_LINE(whitespace/operators)
        CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0,
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data));
#endif
    offset_slice_axis += top_slice_axis;
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0] || top.size() == 1) { return; }
  int offset_slice_axis = 0;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = false;
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
#ifndef USE_FGPU
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top_diff, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, bottom_diff);
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(Slice<Dtype>,  // NOLINT_NEXT_LINE(whitespace/operators)
        CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0,
        nthreads, top_diff, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, bottom_diff));
#endif
    offset_slice_axis += top_slice_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SliceLayer);

}  // namespace caffe
