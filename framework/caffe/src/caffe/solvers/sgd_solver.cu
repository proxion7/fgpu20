#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"

#ifdef USE_FGPU
#include <fractional_gpu_cuda.cuh>
#endif

namespace caffe {

#ifndef USE_FGPU
template <typename Dtype>
__global__ void SGDUpdate(int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    g[i] = h[i] = momentum*h[i] + local_rate*g[i];
  }
}
#else
template <typename Dtype>
__global__ FGPU_DEFINE_KERNEL(SGDUpdate, int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate)   {

  fgpu_dev_ctx_t *ctx;
  dim3 _blockIdx, _gridDim;
  ctx = FGPU_DEVICE_INIT();
  _gridDim = FGPU_GET_GRIDDIM(ctx);

  FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

    CUDA_KERNEL_LOOP(i, N, _blockIdx, _gridDim) {
      Dtype *gaddr = FGPU_COLOR_TRANSLATE_ADDR(ctx, &g[i]);
      Dtype *haddr = FGPU_COLOR_TRANSLATE_ADDR(ctx, &h[i]);

      *gaddr = *haddr = momentum*(*haddr) + local_rate* (*gaddr);
    }

  } 
}
#endif

template <typename Dtype>
void sgd_update_gpu(int N, Dtype* g, Dtype* h, Dtype momentum,
    Dtype local_rate) {
#ifndef USE_FGPU
  SGDUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, momentum, local_rate);
  CUDA_POST_KERNEL_CHECK;
#else
    FGPU_CHECK(FGPU_LAUNCH_KERNEL(SGDUpdate<Dtype>,  // NOLINT_NEXT_LINE(whitespace/operators)
      CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0,
      N, g, h, momentum, local_rate));
#endif
}
template void sgd_update_gpu<float>(int, float*, float*, float, float);
template void sgd_update_gpu<double>(int, double*, double*, double, double);

}  // namespace caffe
