#include "hip/hip_runtime.h"
/* 
 * This file takes care of memory coloring in GPU. To do this, we make 
 * custom ioctl() calls to nvidia uvm driver.  These ioctls have been added to 
 * vanilla uvm driver to expose certain functionality.
 * This file traps some calls made by CUDA library using preload mechanisms.
 * This is needed because CUDA library is closed source.
 */

/* TODO: Use better error codes */
/* TODO: Half of colored memory is being wasted. Need to resolve this issue */
/* 
 * TODO: Make PTEs on GPU consistent (on memprefetch to CPU they are invidated
 * for uvm to work). But make sure data migrates when data changes (When user 
 * explicitly requests)
 */
/* 
 * TODO: There shouldn't be need to memprefetch incase data hasn't changed
 * between CPU and GPU. This should work when GPU TLBs are made persistent.
 * Check what happens currently.
 */

#include <assert.h>
#include <dlfcn.h>
#include <errno.h>
#include <iostream>
#include <inttypes.h>
#include <linux/ioctl.h>
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/ioctl.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <unistd.h>

/* CUDA/NVML */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nvml.h>
#include <hip/driver_types.h>

/* NVIDIA driver */
#include <uvm_minimal_init.h>
#include <nvCpuUuid.h>

#include <fractional_gpu.hpp>
#include <fractional_gpu_cuda.cuh>

#include <fgpu_internal_allocator.hpp>
#include <fgpu_internal_memory.hpp>

#ifdef FGPU_MEM_COLORING_ENABLED

#define NVIDIA_UVM_DEVICE_PATH  "/dev/" NVIDIA_UVM_DEVICE_NAME
/* TODO: This path can be changed via environment variable */
#define NVIDIA_MPS_CONTROL_PATH "/tmp/nvidia-mps/control"

/* Ioctl codes */
#define IOCTL_GET_DEVICE_COLOR_INFO     _IOC(0, 0, UVM_GET_DEVICE_COLOR_INFO, 0)
#define IOCTL_GET_PROCESS_COLOR_INFO    _IOC(0, 0, UVM_GET_PROCESS_COLOR_INFO, 0)
#define IOCTL_SET_PROCESS_COLOR_INFO    _IOC(0, 0, UVM_SET_PROCESS_COLOR_INFO, 0)
#define IOCTL_MEMCPY_COLORED            _IOC(0, 0, UVM_MEMCPY_COLORED, 0)
#define IOCTL_MEMSET_COLORED            _IOC(0, 0, UVM_MEMSET_COLORED, 0)

/* UVM device fd */
static int g_uvm_fd = -1;

typedef int (*orig_open_f_type)(const char *pathname, int flags, int mode);
orig_open_f_type g_orig_open;

typedef int (*orig_connect_f_type)(int sockfd, const struct sockaddr *addr,
                   socklen_t addrlen);
orig_connect_f_type g_orig_connect;

pthread_once_t g_pre_init_once = PTHREAD_ONCE_INIT;
pthread_once_t g_post_init_once = PTHREAD_ONCE_INIT;
bool g_init_failed;

/* All information needed for tracking memory */
struct {
    bool is_initialized;

    /* Start physical address of allocation */
    void *base_phy_addr;

    /* Actual memory available for coloring */
    size_t reserved_len;

    /* Actual memory allocation */
    void *base_addr;

    int color;

    allocator_t *allocator;

} g_memory_ctx;

/* Does the most neccesary initialization */
static void pre_initialization(void)
{
    g_orig_open = (orig_open_f_type)dlsym(RTLD_NEXT,"open");
    if (!g_orig_open) {
        g_init_failed = true;
        return;
    }

    g_orig_connect = (orig_connect_f_type)dlsym(RTLD_NEXT,"connect");
    if (!g_orig_connect) {
        g_init_failed = true;
        return;
    }
}

static void post_initialization(void)
{
    nvmlReturn_t ncode;

    ncode = nvmlInit();
    if (ncode != NVML_SUCCESS) {
        g_init_failed = true;
        return;
    }
}

/* Does the initialization atmost once */
static int init(bool do_post_init)
{
    int ret;

    ret = pthread_once(&g_pre_init_once, pre_initialization);
    if (ret < 0)
        return ret;
    
    if (g_init_failed) {
        fprintf(stderr, "FGPU:Initialization failed\n");
        return -EINVAL;
    }
    
    if (!do_post_init)
        return 0;

    ret = pthread_once(&g_post_init_once, post_initialization);
    if (ret < 0)
        return ret;
    
    if (g_init_failed) {
        fprintf(stderr, "FGPU:Initialization failed\n");
        return -EINVAL;
    }

    return 0;
}

/* Retrieve the device UUID from the CUDA device handle */
static int get_device_UUID(int device, NvProcessorUuid *uuid)
{
    nvmlReturn_t ncode;
    hipError_t ccode;
    char pciID[32];
    nvmlDevice_t handle;
    char buf[100];
    char hex[3];
    char *nbuf;
    int cindex, hindex, uindex, needed_bytes;
    char c;
    int len;
    std::string prefix = "GPU";
    const char *gpu_prefix = prefix.c_str();
    int gpu_prefix_len = strlen(gpu_prefix);

    /* Get PCI ID from the device handle and then use NVML library to get UUID */
    ccode = hipDeviceGetPCIBusId(pciID, sizeof(pciID), device);
    if (ccode != hipSuccess) {
        fprintf(stderr, "FGPU:Couldn't find PCI Bus ID\n");
        return -EINVAL;
    }

    ncode = nvmlDeviceGetHandleByPciBusId(pciID, &handle);
    if (ncode != NVML_SUCCESS){
        fprintf(stderr, "FGPU:Couldn't get Device Handle\n");
        return -EINVAL;
    }

     
    ncode = nvmlDeviceGetUUID(handle, buf, sizeof(buf));
    if (ncode != NVML_SUCCESS){
        fprintf(stderr, "FGPU:Couldn't find device UUID\n");
        return -EINVAL;
    }

    if (strncmp(buf, gpu_prefix, gpu_prefix_len != 0))
        return 0;

    nbuf = buf + gpu_prefix_len;

    /*
     * UUID has characters and hexadecimal numbers. 
     * We are only interested in hexadecimal numbers.
     * Each hexadecimal numbers is equal to 1 byte.
     */
    needed_bytes = sizeof(NvProcessorUuid);
    len = strlen(nbuf);

    for (cindex = 0, hindex = 0, uindex = 0; cindex < len; cindex++) {
        c = nbuf[cindex];
        if ((c >= 'a' && c <= 'z') || (c >= 'A' && c <= 'Z') || (c >= '0' && c <= '9')) {
            hex[hindex] = c;
            hindex++;
            if (hindex == 2) {
                hex[2] = '\0';
                uuid->uuid[uindex] = (uint8_t)strtol(hex, NULL, 16);
                uindex++;
                hindex = 0;
                if (uindex > needed_bytes) {
                    fprintf(stderr, "FGPU:Invalid device UUID\n");
                    return -EINVAL;
                }  
            }
        }
    }

    if (uindex != needed_bytes) {
        fprintf(stderr, "FGPU:Invalid device UUID\n");
        return -EINVAL;
    }

    return 0;
}

extern "C" {

/* Trap open() calls (interested in UVM device opened by CUDA) */
int open(const char *pathname, int flags, int mode)
{
    int ret;

    ret = init(false);
    if (ret < 0)
        return ret;
    
    ret = g_orig_open(pathname,flags, mode);

    if (g_uvm_fd < 0 && 
            strncmp(pathname, NVIDIA_UVM_DEVICE_PATH, strlen(NVIDIA_UVM_DEVICE_PATH)) == 0) {
        g_uvm_fd = ret;
    }

    return ret;
}

/* Trap connect() calls (interested in connection to MPS) */
int connect(int sockfd, const struct sockaddr *addr,
                   socklen_t addrlen)
{
    int ret;

    ret = init(false);
    if (ret < 0)
        return ret;
    
    ret = g_orig_connect(sockfd, addr, addrlen);

    if (ret >= 0 && g_uvm_fd < 0 && addr && addr->sa_family == AF_LOCAL && 
            strncmp(addr->sa_data, NVIDIA_MPS_CONTROL_PATH, strlen(NVIDIA_MPS_CONTROL_PATH)) == 0) {
        g_uvm_fd = sockfd;
    }

    return ret;
}

} /* extern "C" */

static int get_device_color_info(int device, int *num_colors, size_t *max_len)
{
    UVM_GET_DEVICE_COLOR_INFO_PARAMS params;
    int ret;

    ret = get_device_UUID(device, &params.destinationUuid);
    if (ret < 0)
        return ret;
    
    ret = ioctl(g_uvm_fd, IOCTL_GET_DEVICE_COLOR_INFO, &params);
    if (ret < 0)
        return ret;

    if (params.rmStatus != NV_OK) {
        fprintf(stderr, "FGPU:Couldn't get device color info\n");
        return -EINVAL;
    }

    if (num_colors)
        *num_colors = params.numColors;

    if (max_len)
        *max_len = params.maxLength;

    return 0;

}

/* Get the numbers of colors supported by the memory and maximum memory that can be reserved */
int fgpu_memory_get_device_info(int *num_colors, size_t *max_len)
{
    int ret;

    ret = init(true);
    if (ret < 0)
        return ret;

    if (g_uvm_fd < 0) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }

    return get_device_color_info(FGPU_DEVICE_NUMBER, num_colors, max_len);

}

static int get_process_color_info(int device, int *color, size_t *length)
{
    UVM_GET_PROCESS_COLOR_INFO_PARAMS params;
    int ret;

    ret = get_device_UUID(device, &params.destinationUuid);
    if (ret < 0)
        return ret;


    ret = ioctl(g_uvm_fd, IOCTL_GET_PROCESS_COLOR_INFO, &params);
    if (ret < 0)
        return ret;

    if (params.rmStatus != NV_OK) {
        fprintf(stderr, "FGPU:Couldn't get process color property\n");
        return -EINVAL;
    }

    if (color)
        *color = params.color;

    if (length)
        *length = params.length;

    return 0;
}

/* Indicates the color set currently for the process and the length reserved */
int fgpu_process_get_colors_info(int device, int *color, size_t *length)
{
    int ret;

    ret = init(true);
    if (ret < 0)
        return ret;

    if (g_uvm_fd < 0) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }

    return get_process_color_info(device, color, length);
}

/* Set memory color and also reserve memory */
static int set_process_color_info(int device, int color, size_t req_length,
        hipStream_t stream)
{
    UVM_SET_PROCESS_COLOR_INFO_PARAMS params;
    size_t actual_length = req_length;
    int ret;

    /* Color can only be set once */
    if (g_memory_ctx.is_initialized) {
        fprintf(stderr, "FGPU:Process color already set\n");
        return -EINVAL;
    }

#if defined(FGPU_USER_MEM_COLORING_ENABLED)
    int num_colors;
    ret = get_device_color_info(device, &num_colors, NULL);
    if (ret < 0)
        return ret;
    
    actual_length = req_length * num_colors;
#endif

    ret = get_device_UUID(device, &params.destinationUuid);
    if (ret < 0)
        return ret;

    params.color = color;
    params.length = actual_length;

    ret = ioctl(g_uvm_fd, IOCTL_SET_PROCESS_COLOR_INFO, &params);
    if (ret < 0)
        return ret;

    if (params.rmStatus != NV_OK) {
        fprintf(stderr, "FGPU:Couldn't set process color property\n");
        return -EINVAL;
    }

    ret = gpuErrCheck(hipMallocManaged(&g_memory_ctx.base_addr, actual_length));
    if (ret < 0)
        return ret;

    /* Do the actual allocation on device */
    ret = gpuErrCheck(hipMemPrefetchAsync(g_memory_ctx.base_addr, actual_length,
                device, stream));
    if (ret < 0) {
        hipFree(g_memory_ctx.base_addr);
        return ret;
    }

    ret = gpuErrCheck(hipStreamSynchronize(stream));
    if (ret < 0) {
    	hipFree(g_memory_ctx.base_addr);
        return ret;
    }

    g_memory_ctx.is_initialized = true;
    g_memory_ctx.base_phy_addr = (void *)params.address;
    g_memory_ctx.reserved_len = req_length;
    g_memory_ctx.color = color;

    g_memory_ctx.allocator = allocator_init(g_memory_ctx.base_addr, 
            req_length, FGPU_DEVICE_ADDRESS_ALIGNMENT);
    if (!g_memory_ctx.allocator) {
        fprintf(stderr, "FGPU:Allocator Initialization Failed\n");
        return -EINVAL;
    }
    return 0;
}

/* Indicates the color set currently for the process and the length reserved */
int fgpu_memory_set_colors_info(int device, int color, size_t length,
        hipStream_t stream)
{
    int ret;

    ret = init(true);
    if (ret < 0)
        return ret;

    if (g_uvm_fd < 0) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }

    return set_process_color_info(device, color, length, stream);
}

void fgpu_memory_deinit(void)
{
    if (!g_memory_ctx.is_initialized)
        return;

    if (g_memory_ctx.allocator)
        allocator_deinit(g_memory_ctx.allocator);

    hipFree(g_memory_ctx.base_addr);

    g_memory_ctx.is_initialized = false;
}

int fgpu_memory_allocate(void **p, size_t len)
{
    void *ret_addr;

    if (!g_memory_ctx.is_initialized) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }


    ret_addr = allocator_alloc(g_memory_ctx.allocator, len);
    if (!ret_addr) {
        fprintf(stderr, "FGPU:Can't allocate device memory\n");
        return -ENOMEM;
    }

    *p = ret_addr;
    
    return 0;
}

int fgpu_memory_free(void *p)
{
    if (!g_memory_ctx.is_initialized) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }

    allocator_free(g_memory_ctx.allocator, p);

    return 0;
}

/* Useful for only reverse engineering */
void *fgpu_memory_get_phy_address(void *addr)
{
    if (!g_memory_ctx.base_phy_addr)
        return NULL;

    return (void *)((uintptr_t)g_memory_ctx.base_phy_addr + 
            (uintptr_t)addr - (uintptr_t)g_memory_ctx.base_addr);
}


#else /* FGPU_MEM_COLORING_ENABLED */

int fgpu_memory_allocate(void **p, size_t len)
{
    /*
     * XXX: We are using hipMallocManaged() nstead of just
     * hipMalloc() because to make comparision fair between memory coloring
     * enabled v.s. disabled. Memcpy() is slower (for small sizes) for
     * hipMallocManaged() v.s. for hipMalloc() (but faster for larger sizes > 8MB)
     * This we suspect is because of code difference inside the Linux driver
     */
    int ret;
    
    ret = gpuErrCheck(hipMallocManaged(p, len));
    if (ret < 0)
        return ret;

    /* Do the actual allocation on device */
    ret = gpuErrCheck(hipMemPrefetchAsync(*p, len, FGPU_DEVICE_NUMBER));
    if (ret < 0) {
        hipFree(p);
        return ret;
    }

    return gpuErrCheck(hipDeviceSynchronize());
}

int fgpu_memory_free(void *p)
{
    return gpuErrCheck(hipFree(p));
}

void *fgpu_memory_get_phy_address(void *addr)
{
    assert(0);
    return NULL;
}

#endif /* FGPU_MEM_COLORING_ENABLED */

#if defined(FGPU_USER_MEM_COLORING_ENABLED)

int fgpu_get_memory_info(uintptr_t *start_virt_addr, uintptr_t *start_idx)
{
    if (!g_memory_ctx.is_initialized) {
        fprintf(stderr, "FGPU:Initialization not done\n");
        return -EBADF;
    }

    *start_virt_addr = (uintptr_t)g_memory_ctx.base_addr;
    *start_idx = ((uintptr_t)g_memory_ctx.base_phy_addr) >> FGPU_DEVICE_COLOR_SHIFT;

    return 0;
}

/* 
 * TODO: This might be slower to loop in userspace. Doing this inside kernel
 * might be faster. So measure the reduction in bandwidth and if substantial,
 * do inside kernel
 */
int fgpu_memory_copy_async_to_device_internal(void *dst, const void *src, 
                                                size_t count, hipStream_t stream)
{
    size_t left = count;
    int ret;

    while (left) {
        uintptr_t base = (uintptr_t)dst & FGPU_DEVICE_PAGE_MASK;
        uintptr_t offset = (uintptr_t)dst - base;
        size_t transfer = min(min(left, (size_t)FGPU_DEVICE_PAGE_SIZE), 
                (size_t)FGPU_DEVICE_PAGE_SIZE - (size_t)offset);
        void *true_virt_addr_dest = fgpu_color_device_true_virt_addr((uint64_t)g_memory_ctx.base_addr,
                                                                     (uint64_t)g_memory_ctx.base_phy_addr,
                                                                     g_memory_ctx.color,
                                                                     dst);

        ret = gpuErrCheck(hipMemcpyAsync(true_virt_addr_dest, src, transfer, hipMemcpyHostToDevice, stream));
        if (ret < 0)
            return ret;
        dst = (void *)((uintptr_t)dst + transfer);
        src = (void *)((uintptr_t)src + transfer);
        left -= transfer;
    }
    return 0;
}

int fgpu_memory_copy_async_to_host_internal(void *dst, const void *src, 
                                                size_t count, hipStream_t stream)
{
    size_t left = count;
    int ret;

    while (left) {
        uintptr_t base = (uintptr_t)src & FGPU_DEVICE_PAGE_MASK;
        uintptr_t offset = (uintptr_t)src - base;
        size_t transfer = min(min(left, (size_t)FGPU_DEVICE_PAGE_SIZE), 
                (size_t)FGPU_DEVICE_PAGE_SIZE - (size_t)offset);
        void *true_virt_addr_src = fgpu_color_device_true_virt_addr((uint64_t)g_memory_ctx.base_addr,
                                                                    (uint64_t)g_memory_ctx.base_phy_addr,
                                                                    g_memory_ctx.color,
                                                                    src);

        ret = gpuErrCheck(hipMemcpyAsync(dst, true_virt_addr_src, transfer, hipMemcpyDeviceToHost, stream));
        if (ret < 0)
            return ret;
        dst = (void *)((uintptr_t)dst + transfer);
        src = (void *)((uintptr_t)src + transfer);
        left -= transfer;
    }

    return 0;
}

/* Using kernel provided colored memcopy instead of doing it in userspace */
/*
int fgpu_memory_copy_async_internal(void *dst, const void *src, size_t count,
                                    enum fgpu_memory_copy_type type,
                                    hipStream_t stream)
{

    switch (type) {
    case FGPU_COPY_CPU_TO_GPU:
        return fgpu_memory_copy_async_to_device_internal(dst, src, count, stream);
    case FGPU_COPY_GPU_TO_CPU:
        return fgpu_memory_copy_async_to_host_internal(dst, src, count, stream);
    default:
        return -1;
    }   
}
*/

/* Check if given address lies on GPU */
static bool is_address_on_gpu(const void *address)
{

    if ((uintptr_t)address < (uintptr_t)g_memory_ctx.base_addr)
        return false;
    
    if ((uintptr_t)address >= (uintptr_t)g_memory_ctx.base_addr + 
            g_memory_ctx.reserved_len)
        return false;

    return true;
}

int fgpu_memory_copy_async_internal(void *dst, const void *src, size_t count,
                                    enum fgpu_memory_copy_type type,
                                    hipStream_t stream)
{
    /* XXX: Currently, not sure how to use stream? */
    UVM_MEMCPY_COLORED_PARAMS params;
    int ret;

    if (type == FGPU_COPY_CPU_TO_CPU) {
        memcpy(dst, src, count);
        return 0;
    }

    /* Source is GPU? */
    if (type == FGPU_COPY_GPU_TO_CPU || type == FGPU_COPY_GPU_TO_GPU ||
            (type == FGPU_COPY_DEFAULT && is_address_on_gpu(src))) {
        
        ret = get_device_UUID(FGPU_DEVICE_NUMBER, &params.srcUuid);
        if (ret < 0)
            return ret;
        
        params.srcBase = (NvU64)fgpu_color_device_true_virt_addr((uint64_t)g_memory_ctx.base_addr,
                                                                    (uint64_t)g_memory_ctx.base_phy_addr,
                                                                    g_memory_ctx.color,
                                                                    src);

    } else {
        memcpy(&params.srcUuid, &NV_PROCESSOR_UUID_CPU_DEFAULT, sizeof(NvProcessorUuid));
        params.srcBase = (NvU64)src;
    }

    /* Destination is GPU? */
    if (type == FGPU_COPY_CPU_TO_GPU || type == FGPU_COPY_GPU_TO_GPU ||
            (type == FGPU_COPY_DEFAULT && is_address_on_gpu(dst))) {
        ret = get_device_UUID(FGPU_DEVICE_NUMBER, &params.destUuid);
        if (ret < 0)
            return ret;

        params.destBase = (NvU64)fgpu_color_device_true_virt_addr((uint64_t)g_memory_ctx.base_addr,
                                                                    (uint64_t)g_memory_ctx.base_phy_addr,
                                                                    g_memory_ctx.color,
                                                                    dst);

    } else {
        memcpy(&params.destUuid, &NV_PROCESSOR_UUID_CPU_DEFAULT, sizeof(NvProcessorUuid));
        params.destBase = (NvU64)dst;
    }

    params.length = count;

    ret = ioctl(g_uvm_fd, IOCTL_MEMCPY_COLORED, &params);
    if (ret < 0)
        return ret;

    if (params.rmStatus != NV_OK) {
        fprintf(stderr, "FGPU:Memcpy failed\n");
        return -EINVAL;
    }

    return 0;
}

int fgpu_memory_memset_async_internal(void *address, int value, size_t count, hipStream_t stream)
{
    /* XXX: Currently, not sure how to use stream? */
    UVM_MEMSET_COLORED_PARAMS params;
    int ret;

    ret = get_device_UUID(FGPU_DEVICE_NUMBER, &params.uuid);
    if (ret < 0)
        return ret;
        
    params.base = (NvU64)fgpu_color_device_true_virt_addr((uint64_t)g_memory_ctx.base_addr,
                                                          (uint64_t)g_memory_ctx.base_phy_addr,
                                                          g_memory_ctx.color,
                                                          address);
    params.value = value;
    params.length = count;

    ret = ioctl(g_uvm_fd, IOCTL_MEMSET_COLORED, &params);
    if (ret < 0)
        return ret;

    if (params.rmStatus != NV_OK) {
        fprintf(stderr, "FGPU:Memcpy failed\n");
        return -EINVAL;
    } 

    return 0;
}

#else /* FGPU_USER_MEM_COLORING_ENABLED */

int fgpu_memory_copy_async_internal(void *dst, const void *src, size_t count, enum fgpu_memory_copy_type type, hipStream_t stream)
{
    switch (type) {
    case FGPU_COPY_CPU_TO_GPU:
        return gpuErrCheck(hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream));
    case FGPU_COPY_GPU_TO_CPU:
        return gpuErrCheck(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream));
    case FGPU_COPY_GPU_TO_GPU:
        return gpuErrCheck(hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, stream));
    case FGPU_COPY_CPU_TO_CPU:
        return gpuErrCheck(hipMemcpyAsync(dst, src, count, hipMemcpyHostToHost, stream));
    case FGPU_COPY_DEFAULT:
        return gpuErrCheck(hipMemcpyAsync(dst, src, count, hipMemcpyDefault, stream));
    default:
        assert(0);
        return -1;
    }   
}

int fgpu_memory_memset_async_internal(void *address, int value, size_t count, hipStream_t stream)
{
    return gpuErrCheck(hipMemsetAsync(address, value, count, stream));
}

#endif /* FGPU_USER_MEM_COLORING_ENABLED */
