#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>

#include <fractional_gpu.hpp>
#include <fractional_gpu_cuda.cuh>

#define USE_FGPU
#include <fractional_gpu_testing.hpp>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> 
FGPU_DEFINE_KERNEL(matrixMulCUDA, float *C, float *A, float *B, int wA, int wB)
{
    fgpu_dev_ctx_t *ctx;
    dim3 _blockIdx;
    ctx = FGPU_DEVICE_INIT();

    FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {
        // Block index
        int bx = _blockIdx.x;
        int by = _blockIdx.y;

        // Thread index
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        // Index of the first sub-matrix of A processed by the block
        int aBegin = wA * BLOCK_SIZE * by;

        // Index of the last sub-matrix of A processed by the block
        int aEnd   = aBegin + wA - 1;

       // Step size used to iterate through the sub-matrices of A
        int aStep  = BLOCK_SIZE;

        // Index of the first sub-matrix of B processed by the block
        int bBegin = BLOCK_SIZE * bx;

        // Step size used to iterate through the sub-matrices of B
        int bStep  = BLOCK_SIZE * wB;

        // Csub is used to store the element of the block sub-matrix
        // that is computed by the thread
        float Csub = 0;

        // Loop over all the sub-matrices of A and B
        // required to compute the block sub-matrix
        for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep)
        {

            // Declaration of the shared memory array As used to
            // store the sub-matrix of A
            __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

            // Declaration of the shared memory array Bs used to
            // store the sub-matrix of B
            __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

            // Load the matrices from device memory
            // to shared memory; each thread loads
            // one element of each matrix
            As[ty][tx] = FGPU_COLOR_LOAD(ctx, &A[a + wA * ty + tx]);
            Bs[ty][tx] = FGPU_COLOR_LOAD(ctx, &B[b + wB * ty + tx]);

            // Synchronize to make sure the matrices are loaded
            __syncthreads();

            // Multiply the two matrices together;
            // each thread computes one element
            // of the block sub-matrix
#pragma unroll

            for (int k = 0; k < BLOCK_SIZE; ++k)
            {
                Csub += As[ty][k] * Bs[k][tx];
            }

            // Synchronize to make sure that the preceding
            // computation is done before loading two new
            // sub-matrices of A and B in the next iteration
            __syncthreads();
        }

        // Write the block sub-matrix to device memory;
        // each thread writes one element
        int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
        FGPU_COLOR_STORE(ctx, &C[c + wB * ty + tx], Csub);
    } FGPU_FOR_EACH_END;
}



void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}


/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int num_iterations)
{
    int block_size = 32;
    dim3 dimsA(10 * block_size, 10 * block_size, 1);
    dim3 dimsB(20 * block_size, 10 * block_size, 1);

    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }


    int ret;
    
    ret = fgpu_memory_allocate((void **) &d_A, mem_size_A);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_A returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_allocate((void **) &d_B, mem_size_B);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_B returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_allocate((void **) &d_C, mem_size_C);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_C returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    ret = fgpu_memory_copy_async(d_A, h_A, mem_size_A, FGPU_COPY_CPU_TO_GPU);
    if (ret < 0) {
        printf("fgpu_memory_copy_async h_A returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_copy_async(d_B, h_B, mem_size_B, FGPU_COPY_CPU_TO_GPU);
    if (ret < 0) {
         printf("fgpu_memory_copy_async h_A returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
    {
        printf("fgpu_color_stream_synchronize returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }


    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Execute the kernel
    int nIter = num_iterations;

    double start, total;
    pstats_t stats;

    // Init
    for (int j = 0; j < nIter; j++)
    {
        start = dtime_usec(0);

        if (block_size == 16)
        {
            ret = FGPU_LAUNCH_KERNEL(matrixMulCUDA<16>, grid, threads, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            ret = FGPU_LAUNCH_KERNEL(matrixMulCUDA<32>, grid, threads, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (ret < 0)
            return ret;
	
	    ret = fgpu_color_stream_synchronize();
    	if (ret < 0)
        	return ret;

        total = dtime_usec(start);
        dprintf("Time:%f, BlockSize:%d, dimA.x:%d, dimA.y:%d, dimB.x:%d, dimB.y:%d\n", total, block_size, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
    }

    pstats_init(&stats);
    start = dtime_usec(0);
    for (int j = 0; j < nIter; j++)
    {
        double sub_start = dtime_usec(0);
        if (block_size == 16)
        {
            ret = FGPU_LAUNCH_KERNEL(matrixMulCUDA<16>, grid, threads, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            ret = FGPU_LAUNCH_KERNEL(matrixMulCUDA<32>, grid, threads, 0, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (ret < 0)
            return ret;
        pstats_add_observation(&stats, dtime_usec(sub_start));
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
        return ret;

    total = dtime_usec(start);
    pstats_print(&stats);

    // Compute and print the performance
    double msecPerMatrixMul = total / nIter / 1000;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.6f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);


    // Copy result from device to host
    ret = fgpu_memory_copy_async(h_C, d_C, mem_size_C, FGPU_COPY_GPU_TO_CPU);
    if (ret < 0) {
         printf("fgpu_memory_copy_async h_A returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
    {
        printf("fgpu_color_stream_synchronize returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE        );
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    fgpu_memory_free(d_A);
    fgpu_memory_free(d_B);
    fgpu_memory_free(d_C);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

int main(int argc, char *argv[])
{
    int ret;
    int num_iterations;

    test_initialize(argc, argv, &num_iterations);


    ret = matrixMultiply(num_iterations);
    if (ret < 0)
        return ret;

    test_deinitialize();

    return 0;
}
