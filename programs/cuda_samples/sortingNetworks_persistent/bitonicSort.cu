#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



//Based on http://www.iti.fh-flensburg.de/lang/algorithmen/sortieren/bitonic/bitonicen.htm



#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include "sortingNetworks_common.h"
#include "sortingNetworks_common.cuh"

#include <fractional_gpu.hpp>
#include <fractional_gpu_cuda.cuh>

////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__
FGPU_DEFINE_KERNEL(bitonicSortShared,
    uint *c_d_DstKey,	
    uint *c_d_DstVal,
    uint *c_d_SrcKey,
    uint *c_d_SrcVal,
    uint arrayLength,
    uint dir
)
{
    fgpu_dev_ctx_t *ctx;
   	dim3 _blockIdx;
    
    ctx = FGPU_DEVICE_INIT();

   	FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

        uint *d_DstKey = c_d_DstKey;
        uint *d_DstVal = c_d_DstVal;
        uint *d_SrcKey = c_d_SrcKey;
        uint *d_SrcVal = c_d_SrcVal;

        // Handle to thread block group
		//cg::thread_block cta = cg::this_thread_block();
		//Shared memory storage for one or more short vectors
		__shared__ uint s_key[SHARED_SIZE_LIMIT];
		__shared__ uint s_val[SHARED_SIZE_LIMIT];

		//Offset to the beginning of subbatch and load data
		d_SrcKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_SrcVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		s_key[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[                      0]);
		s_val[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[                      0]);
		s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[(SHARED_SIZE_LIMIT / 2)]);
		s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[(SHARED_SIZE_LIMIT / 2)]);

		for (uint size = 2; size < arrayLength; size <<= 1)
		{
            //Bitonic merge
            uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

            for (uint stride = size / 2; stride > 0; stride >>= 1)
            {

                __syncthreads();
                //cg::sync(cta);
                uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
                Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
                );
            }
		}

		//ddd == dir for the last bitonic merge step
		{
            for (uint stride = arrayLength / 2; stride > 0; stride >>= 1)
            {
                __syncthreads();
                //cg::sync(cta);
                uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
                Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                dir
                );
            }
		}

		//cg::sync(cta);
        __syncthreads();
		FGPU_COLOR_STORE(ctx, &d_DstKey[                      0], s_key[threadIdx.x +                       0]);
		FGPU_COLOR_STORE(ctx, &d_DstVal[                      0], s_val[threadIdx.x +                       0]);
		FGPU_COLOR_STORE(ctx, &d_DstKey[(SHARED_SIZE_LIMIT / 2)], s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
		FGPU_COLOR_STORE(ctx, &d_DstVal[(SHARED_SIZE_LIMIT / 2)], s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
    } FGPU_FOR_EACH_END
}



////////////////////////////////////////////////////////////////////////////////
// Bitonic sort kernel for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
//Bottom-level bitonic sort
//Almost the same as bitonicSortShared with the exception of
//even / odd subarrays being sorted in opposite directions
//Bitonic merge accepts both
//Ascending | descending or descending | ascending sorted pairs
__global__
FGPU_DEFINE_KERNEL(bitonicSortShared1,
    uint *c_d_DstKey,
    uint *c_d_DstVal,
    uint *c_d_SrcKey,
    uint *c_d_SrcVal
)
{
    fgpu_dev_ctx_t *ctx;
   	dim3 _blockIdx;
	ctx = FGPU_DEVICE_INIT();

   	FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

        uint *d_DstKey = c_d_DstKey;
        uint *d_DstVal = c_d_DstVal;
        uint *d_SrcKey = c_d_SrcKey;
        uint *d_SrcVal = c_d_SrcVal;

		// Handle to thread block group
		cg::thread_block cta = cg::this_thread_block();
		//Shared memory storage for current subarray
		__shared__ uint s_key[SHARED_SIZE_LIMIT];
		__shared__ uint s_val[SHARED_SIZE_LIMIT];

		//Offset to the beginning of subarray and load data
		d_SrcKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_SrcVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		d_DstVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
		s_key[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[                      0]);
		s_val[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[                      0]);
		s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[(SHARED_SIZE_LIMIT / 2)]);
		s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[(SHARED_SIZE_LIMIT / 2)]);

		for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1)
		{
		//Bitonic merge
		uint ddd = (threadIdx.x & (size / 2)) != 0;

		for (uint stride = size / 2; stride > 0; stride >>= 1)
		{
		    cg::sync(cta);
		    uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
		    Comparator(
			s_key[pos +      0], s_val[pos +      0],
			s_key[pos + stride], s_val[pos + stride],
			ddd
		    );
		}
		}

		//Odd / even arrays of SHARED_SIZE_LIMIT elements
		//sorted in opposite directions
		uint ddd = _blockIdx.x & 1;
		{
		for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
		{
		    cg::sync(cta);
		    uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
		    Comparator(
			s_key[pos +      0], s_val[pos +      0],
			s_key[pos + stride], s_val[pos + stride],
			ddd
		    );
		}
		}


		cg::sync(cta);
		FGPU_COLOR_STORE(ctx, &d_DstKey[                      0], s_key[threadIdx.x +                       0]);
		FGPU_COLOR_STORE(ctx, &d_DstVal[                      0], s_val[threadIdx.x +                       0]);
		FGPU_COLOR_STORE(ctx, &d_DstKey[(SHARED_SIZE_LIMIT / 2)], s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
		FGPU_COLOR_STORE(ctx, &d_DstVal[(SHARED_SIZE_LIMIT / 2)], s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
	} FGPU_FOR_EACH_END
}

//Bitonic merge iteration for stride >= SHARED_SIZE_LIMIT
__global__
FGPU_DEFINE_KERNEL(bitonicMergeGlobal,
    uint *c_d_DstKey,
    uint *c_d_DstVal,
    uint *c_d_SrcKey,
    uint *c_d_SrcVal,
    uint arrayLength,
    uint size,
    uint stride,
    uint dir
)
{
    fgpu_dev_ctx_t *ctx;
   	dim3 _blockIdx;
	ctx = FGPU_DEVICE_INIT();

   	FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

        uint *d_DstKey = c_d_DstKey;
        uint *d_DstVal = c_d_DstVal;
        uint *d_SrcKey = c_d_SrcKey;
        uint *d_SrcVal = c_d_SrcVal;

	    uint global_comparatorI = _blockIdx.x * blockDim.x + threadIdx.x;
	    uint        comparatorI = global_comparatorI & (arrayLength / 2 - 1);

	    //Bitonic merge
	    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
	    uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

	    uint keyA = FGPU_COLOR_LOAD(ctx, &d_SrcKey[pos +      0]);
	    uint valA = FGPU_COLOR_LOAD(ctx, &d_SrcVal[pos +      0]);
	    uint keyB = FGPU_COLOR_LOAD(ctx, &d_SrcKey[pos + stride]);
	    uint valB = FGPU_COLOR_LOAD(ctx, &d_SrcVal[pos + stride]);

	    Comparator(
		keyA, valA,
		keyB, valB,
		ddd
	    );

	    FGPU_COLOR_STORE(ctx, &d_DstKey[pos +      0], keyA);
	    FGPU_COLOR_STORE(ctx, &d_DstVal[pos +      0], valA);
	    FGPU_COLOR_STORE(ctx, &d_DstKey[pos + stride], keyB);
	    FGPU_COLOR_STORE(ctx, &d_DstVal[pos + stride], valB);
	} FGPU_FOR_EACH_END
}

//Combined bitonic merge steps for
//size > SHARED_SIZE_LIMIT and stride = [1 .. SHARED_SIZE_LIMIT / 2]
__global__
FGPU_DEFINE_KERNEL(bitonicMergeShared,
    uint *c_d_DstKey,
    uint *c_d_DstVal,
    uint *c_d_SrcKey,
    uint *c_d_SrcVal,
    uint arrayLength,
    uint size,
    uint dir
)
{
    fgpu_dev_ctx_t *ctx;
   	dim3 _blockIdx;
	ctx = FGPU_DEVICE_INIT();

   	FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {

        uint *d_DstKey = c_d_DstKey;
        uint *d_DstVal = c_d_DstVal;
        uint *d_SrcKey = c_d_SrcKey;
        uint *d_SrcVal = c_d_SrcVal;

	    // Handle to thread block group
	    cg::thread_block cta = cg::this_thread_block();
	    //Shared memory storage for current subarray
	    __shared__ uint s_key[SHARED_SIZE_LIMIT];
	    __shared__ uint s_val[SHARED_SIZE_LIMIT];

	    d_SrcKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
	    d_SrcVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
	    d_DstKey += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
	    d_DstVal += _blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
	    s_key[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[                      0]);
	    s_val[threadIdx.x +                       0] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[                      0]);
	    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcKey[(SHARED_SIZE_LIMIT / 2)]);
	    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = FGPU_COLOR_LOAD(ctx, &d_SrcVal[(SHARED_SIZE_LIMIT / 2)]);

	    //Bitonic merge
	    uint comparatorI = UMAD(_blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
	    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);

	    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
	    {
            cg::sync(cta);
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
	    }

	    cg::sync(cta);
	    FGPU_COLOR_STORE(ctx, &d_DstKey[                      0], s_key[threadIdx.x +                       0]);
	    FGPU_COLOR_STORE(ctx, &d_DstVal[                      0], s_val[threadIdx.x +                       0]);
	    FGPU_COLOR_STORE(ctx, &d_DstKey[(SHARED_SIZE_LIMIT / 2)], s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
	    FGPU_COLOR_STORE(ctx, &d_DstVal[(SHARED_SIZE_LIMIT / 2)], s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)]);
	} FGPU_FOR_EACH_END
}


////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Helper function (also used by odd-even merge sort)
extern "C" uint factorRadix2(uint *log2L, uint L)
{
    if (!L)
    {
        *log2L = 0;
        return 0;
    }
    else
    {
        for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++);

        return L;
    }
}

extern "C" uint bitonicSort(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint batchSize,
    uint arrayLength,
    uint dir
)
{
    //Nothing to sort
    if (arrayLength < 2)
        return 0;

    //Only power-of-two array lengths are supported by this implementation
    uint log2L;
    uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
    assert(factorizationRemainder == 1);

    dir = (dir != 0);

    uint  blockCount = batchSize * arrayLength / SHARED_SIZE_LIMIT;
    uint threadCount = SHARED_SIZE_LIMIT / 2;
    dim3 threads(threadCount, 1, 1);
    dim3 grid(blockCount, 1, 1);

    if (arrayLength <= SHARED_SIZE_LIMIT)
    {
        assert((batchSize * arrayLength) % SHARED_SIZE_LIMIT == 0);
        FGPU_LAUNCH_KERNEL(bitonicSortShared, grid, threads, 0, d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, dir);
    }
    else
    {
	    FGPU_LAUNCH_KERNEL(bitonicSortShared1, grid, threads, 0, d_DstKey, d_DstVal, d_SrcKey, d_SrcVal);

        for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
            for (unsigned stride = size / 2; stride > 0; stride >>= 1)
                if (stride >= SHARED_SIZE_LIMIT)
                {
		            threads.x = 256;
		            grid.x = (batchSize * arrayLength) / 512;
                    FGPU_LAUNCH_KERNEL(bitonicMergeGlobal, grid, threads, 0, d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride, dir);
                }
                else
                {
                    threads.x = threadCount;
		            grid.x = blockCount;
		            FGPU_LAUNCH_KERNEL(bitonicMergeShared, grid, threads, 0, d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, dir);
                    break;
                }
    }

    return threadCount;
}
