#include "hip/hip_runtime.h"
// Note: Most of the code comes from the MacResearch OpenCL podcast

#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <fractional_gpu_testing.hpp>

#include "bmp.h"

__global__ void render(char *out, int width, int height) {
  unsigned int x_dim = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y_dim = blockIdx.y*blockDim.y + threadIdx.y;
  int index = 3*width*y_dim + x_dim*3;
  float x_origin = ((float) x_dim/width)*3.25 - 2;
  float y_origin = ((float) y_dim/width)*2.5 - 1.25;

  float x = 0.0;
  float y = 0.0;

  int iteration = 0;
  int max_iteration = 256;
  while(x*x + y*y <= 4 && iteration < max_iteration) {
    float xtemp = x*x - y*y + x_origin;
    y = 2*x*y + y_origin;
    x = xtemp;
    iteration++;
  }

  if(iteration == max_iteration) {
    out[index] = 0;
    out[index + 1] = 0;
    out[index + 2] = 0;
  } else {
    out[index] = iteration;
    out[index + 1] = iteration;
    out[index + 2] = iteration;
  }
}

void runCUDA(int width, int height, int nIter)
{
  // Multiply by 3 here, since we need red, green and blue for each pixel
  size_t buffer_size = sizeof(char) * width * height * 3;

  char *image;
  double start, total;

  hipMalloc((void **) &image, buffer_size);

  char *host_image = (char *) malloc(buffer_size);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(width / blockDim.x, height / blockDim.y, 1);
  
  start = dtime_usec(0);
  render<<< gridDim, blockDim, 0 >>>(image, width, height);
  hipDeviceSynchronize();
  total = dtime_usec(start);

  printf("Time:%f us\n", total);
  printf("Looping\n");

  start = dtime_usec(0);
  for (int i = 0; i < nIter; i++) {
    render<<< gridDim, blockDim, 0 >>>(image, width, height);
  }
  hipDeviceSynchronize();
  total = dtime_usec(start);

  printf("Avg Time:%f us\n", total / nIter);

  hipMemcpy(host_image, image, buffer_size, hipMemcpyDeviceToHost);

  // Now write the file
  write_bmp("output.bmp", width, height, host_image);

  hipFree(image);
  free(host_image);
}

int main(int argc, char **argv) {

  int num_iterations;

  test_initialize(argc, argv, &num_iterations);

  runCUDA(4096, 4096, num_iterations);

  test_deinitialize();
  return 0;
}
