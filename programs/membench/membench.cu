/* This program checks the memory bandwidth */
#include <stdio.h>
#include <assert.h>

#include <fractional_gpu.hpp>

#include <fractional_gpu_testing.hpp>

#define N                   (128 * 1024 * 1024)
#define PAGE_SIZE           (4 * 1024)
#define LARGE_PAGE_SIZE     (2 * 1024 * 1024)
#define HUGE_PAGE_SIZE      (128 * 1024 * 1024)

size_t page_sizes[] = {PAGE_SIZE, LARGE_PAGE_SIZE, HUGE_PAGE_SIZE, HUGE_PAGE_SIZE, LARGE_PAGE_SIZE, PAGE_SIZE};

void transfer_one(void *dst, void *src, size_t chunk, hipMemcpyKind kind, hipStream_t stream)
{
    for (int i = 0; i < N; i += chunk) {
        gpuErrAssert(hipMemcpyAsync(dst, src, chunk, kind, stream));
        src = (void *)((uintptr_t)src + chunk);
        dst = (void *)((uintptr_t)dst + chunk);
    }
}

void transfer_two(void *dst1, void *src1, void *dst2, void *src2, size_t chunk,
        hipMemcpyKind kind1, hipMemcpyKind kind2,
        hipStream_t stream1, hipStream_t stream2)
{
    for (int i = 0; i < N; i += chunk) {
        gpuErrAssert(hipMemcpyAsync(dst1, src1, chunk, kind1, stream1));
        src1 = (void *)((uintptr_t)src1 + chunk);
        dst1 = (void *)((uintptr_t)dst1 + chunk);
        gpuErrAssert(hipMemcpyAsync(dst2, src2, chunk, kind2, stream2));
        src2 = (void *)((uintptr_t)src2 + chunk);
        dst2 = (void *)((uintptr_t)dst2 + chunk);
    }
}

void transfer_managed_one(void *ptr, size_t chunk, int dstDevice, hipStream_t stream)
{
    for (int i = 0; i < N; i += chunk) {
        gpuErrAssert(hipMemPrefetchAsync(ptr, chunk, dstDevice, stream));
        ptr = (void *)((uintptr_t)ptr + chunk);
    }
}
double bandwidth(double time)
{
    return ((double)N) / time / 1000;
}
int main(int argc, char *argv[])
{
    char *x, *y, *h_x, *h_y, *d_x, *d_y, *m_x, *m_y;
    hipStream_t stream_x;
    hipStream_t stream_y;
    double start;

    int num_iterations;

    test_initialize(argc, argv, &num_iterations);


    gpuErrAssert(hipStreamCreate(&stream_x));
    gpuErrAssert(hipStreamCreate(&stream_y));

    x = (char *)malloc(N*sizeof(char));
    y = (char *)malloc(N*sizeof(char));
    assert(x);
    assert(y);

    gpuErrAssert(hipHostAlloc(&h_x, N*sizeof(char), hipHostMallocDefault));
    gpuErrAssert(hipHostAlloc(&h_y, N*sizeof(char), hipHostMallocDefault));

    gpuErrAssert(hipMalloc(&d_x, N*sizeof(char)));
    gpuErrAssert(hipMalloc(&d_y, N*sizeof(char)));

    gpuErrAssert(hipMallocManaged(&m_x, N*sizeof(char)));
    gpuErrAssert(hipMallocManaged(&m_y, N*sizeof(char)));

    // Warmup
    printf("Doing Warmup\n");
    for (int i = 0; i < sizeof(page_sizes) / sizeof(page_sizes[0]); i++) {
        size_t page_size = page_sizes[i];

        /* Test one way transfer */
        transfer_one(d_x, x, page_size, hipMemcpyHostToDevice, stream_x);
        gpuErrAssert(hipStreamSynchronize(stream_x));

        transfer_one(y, d_y, page_size, hipMemcpyDeviceToHost, stream_y);
        gpuErrAssert(hipStreamSynchronize(stream_y));

        if (page_size != PAGE_SIZE) {
            transfer_managed_one(m_x, page_size, 0, stream_x);
            gpuErrAssert(hipStreamSynchronize(stream_x));
        }   
    }

    printf("Warmup done\n");

    for (int i = 0; i < sizeof(page_sizes) / sizeof(page_sizes[0]); i++) {
        size_t page_size = page_sizes[i];
    
        /* Test one way transfer */
        start = dtime_usec(0);
        transfer_one(d_x, x, page_size, hipMemcpyHostToDevice, stream_x);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        printf("HostToDevice: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));

        start = dtime_usec(0);
        transfer_one(x, d_x, page_size, hipMemcpyDeviceToHost, stream_x);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        printf("DeviceToHost: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));

        start = dtime_usec(0);
        transfer_one(d_x, h_x, page_size, hipMemcpyHostToDevice, stream_x);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        printf("HostToDevicePinned: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));

        start = dtime_usec(0);
        transfer_one(h_x, d_x, page_size, hipMemcpyDeviceToHost, stream_x);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        printf("DeviceToHostPinned: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));


        start = dtime_usec(0);
        transfer_two(d_x, x, y, d_y, page_size, hipMemcpyHostToDevice, hipMemcpyDeviceToHost, stream_x, stream_y);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        gpuErrAssert(hipStreamSynchronize(stream_y));
        printf("BothDirections: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));

        start = dtime_usec(0);
        transfer_two(h_x, x, y, h_y, page_size, hipMemcpyHostToDevice, hipMemcpyDeviceToHost, stream_x, stream_y);
        gpuErrAssert(hipStreamSynchronize(stream_x));
        gpuErrAssert(hipStreamSynchronize(stream_y));
        printf("BothDirectionsPinned: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));

        // Too slow with page size 
        if (page_size != PAGE_SIZE) {
        
            // Modify data;
            for (int i = 0; i < N; i += PAGE_SIZE)
                m_x[i] = 0;

            start = dtime_usec(0);
            transfer_managed_one(m_x, page_size, 0, stream_x);
            gpuErrAssert(hipStreamSynchronize(stream_x));
            printf("MemprefetchToDevice: PageSize:%zu, Bandwidth:%f GB/s\n", page_size, bandwidth(dtime_usec(start)));
        }

        printf("\n\n");
    }

    free(x);
    free(y);
    hipHostFree(h_x);
    hipHostFree(h_y);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(m_x);
    hipFree(m_y);

    test_deinitialize();
}
