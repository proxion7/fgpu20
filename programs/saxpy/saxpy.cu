#include "hip/hip_runtime.h"
#include <stdio.h>

#include <fractional_gpu_testing.hpp>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char **argv)
{
  int N = 1<<20;
  int nIter;
  pstats_t stats;

  test_initialize(argc, argv, &nIter);
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Functional test
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  if (maxError != 0) {
      fprintf(stderr, "Failed: Error too large\n");
      exit(-1);
  }

  // Warmup
  for (int i = 0; i < nIter; i++) {

    double sub_start = dtime_usec(0);
        
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    hipDeviceSynchronize();

    dprintf("Time:%f\n", dtime_usec(sub_start));
  }

  // Actual
  pstats_init(&stats);
  for (int j = 0; j < nIter; j++)
  {
    double sub_start = dtime_usec(0);
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    hipDeviceSynchronize();
    pstats_add_observation(&stats, dtime_usec(sub_start));
  }
    
  hipDeviceSynchronize();

  pstats_print(&stats);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  test_deinitialize();
}
